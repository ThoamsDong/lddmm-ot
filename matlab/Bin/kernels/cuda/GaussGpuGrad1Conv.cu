#include "hip/hip_runtime.h"
// Author : B. Charlier (2017)

#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <mex.h>


#define UseCudaOnDoubles USE_DOUBLE_PRECISION

///////////////////////////////////////
///// GRAD CONV ///////////////////////
///////////////////////////////////////


	template < typename TYPE, int DIMPOINT, int DIMVECT >
__global__ void GaussGpuGrad1ConvOnDevice(TYPE ooSigma2,
        TYPE *alpha, TYPE *x, TYPE *y, TYPE *beta, TYPE *gamma,
        int nx, int ny)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // the following line does not work with nvcc 3.0 (it is a bug; it works with anterior and posterior versions)
    // extern __shared__ TYPE SharedData[];  // shared data will contain x and alpha data for the block
    // here is the bug fix (see http://forums.nvidia.com/index.php?showtopic=166905)
    extern __shared__ char SharedData_char[];
    TYPE* const SharedData = reinterpret_cast<TYPE*>(SharedData_char);
    // end of bug fix

    TYPE xi[DIMPOINT], alphai[DIMVECT], xmy[DIMPOINT], gammai[DIMPOINT];
    if(i<nx)  // we will compute gammai only if i is in the range
    {
        // load xi and alphai from device global memory
        for(int k=0; k<DIMPOINT; k++)
            xi[k] = x[i*DIMPOINT+k];
        for(int k=0; k<DIMVECT; k++)
            alphai[k] = alpha[i*DIMVECT+k];
        for(int k=0; k<DIMPOINT; k++)
            gammai[k] = 0.0f;
    }

    for(int jstart = 0, tile = 0; jstart < ny; jstart += blockDim.x, tile++)
    {
        int j = tile * blockDim.x + threadIdx.x;
        if(j<ny) // we load yj and betaj from device global memory only if j<ny
        {
            int inc = DIMPOINT + DIMVECT;
            for(int k=0; k<DIMPOINT; k++)
                SharedData[threadIdx.x*inc+k] = y[j*DIMPOINT+k];
            for(int k=0; k<DIMVECT; k++)
                SharedData[threadIdx.x*inc+DIMPOINT+k] = beta[j*DIMVECT+k];
        }
        __syncthreads();
        if(i<nx) // we compute gammai only if i is in the range
        {
            TYPE *yj, *betaj;
            yj = SharedData;
            betaj = SharedData + DIMPOINT;
            int inc = DIMPOINT + DIMVECT;
            for(int jrel = 0; jrel < blockDim.x && jrel<ny-jstart; jrel++, yj+=inc, betaj+=inc)
            {
                TYPE r2 = 0.0f, sga = 0.0f;
                for(int k=0; k<DIMPOINT; k++)
                {
                    xmy[k] =  xi[k]-yj[k];
                    r2 += xmy[k]*xmy[k];
                }
                for(int k=0; k<DIMVECT; k++)
                    sga += betaj[k]*alphai[k];
                TYPE s =  (-ooSigma2*2.0f*sga) * exp(-r2*ooSigma2);
                for(int k=0; k<DIMPOINT; k++)
                    gammai[k] += s * xmy[k];
            }
        }
        __syncthreads();
    }

    // Save the result in global memory.
    if(i<nx)
        for(int k=0; k<DIMPOINT; k++)
            gamma[i*DIMPOINT+k] = gammai[k];
}

//////////////////////////////////////////////////////////////

extern "C" int GaussGpuGrad1Conv_float(float ooSigma2,
                                       float* alpha_h, float* x_h, float* y_h, float* beta_h, float* gamma_h,
                                       int dimPoint, int dimVect, int nx, int ny)
{

    // Data on the device.
    float* x_d;
    float* y_d;
    float* alpha_d;
    float* gamma_d;
    float* beta_d;

    // Allocate arrays on device.
    hipMalloc((void**)&x_d, sizeof(float)*(nx*dimPoint));
    hipMalloc((void**)&y_d, sizeof(float)*(ny*dimPoint));
    hipMalloc((void**)&alpha_d, sizeof(float)*(nx*dimVect));
    hipMalloc((void**)&beta_d, sizeof(float)*(ny*dimVect));
    hipMalloc((void**)&gamma_d, sizeof(float)*(nx*dimPoint));

    // Send data from host to device.
    hipMemcpy(x_d, x_h, sizeof(float)*(nx*dimPoint), hipMemcpyHostToDevice);
    hipMemcpy(y_d, y_h, sizeof(float)*(ny*dimPoint), hipMemcpyHostToDevice);
    hipMemcpy(alpha_d, alpha_h, sizeof(float)*(nx*dimVect), hipMemcpyHostToDevice);
    hipMemcpy(beta_d, beta_h, sizeof(float)*(ny*dimVect), hipMemcpyHostToDevice);

    // compute on device.
    dim3 blockSize;
    blockSize.x = CUDA_BLOCK_SIZE; // number of threads in each block
    dim3 gridSize;
    gridSize.x =  nx / blockSize.x + (nx%blockSize.x==0 ? 0 : 1);

    if(dimPoint==1 && dimVect==1)
        GaussGpuGrad1ConvOnDevice<float,1,1><<<gridSize,blockSize,blockSize.x*(dimPoint+dimVect)*sizeof(float)>>>
        (ooSigma2, alpha_d, x_d, y_d, beta_d, gamma_d, nx, ny);
    else if(dimPoint==2 && dimVect==1)
        GaussGpuGrad1ConvOnDevice<float,2,1><<<gridSize,blockSize,blockSize.x*(dimPoint+dimVect)*sizeof(float)>>>
        (ooSigma2, alpha_d, x_d, y_d, beta_d, gamma_d, nx, ny);
    else if(dimPoint==2 && dimVect==2)
        GaussGpuGrad1ConvOnDevice<float,2,2><<<gridSize,blockSize,blockSize.x*(dimPoint+dimVect)*sizeof(float)>>>
        (ooSigma2, alpha_d, x_d, y_d, beta_d, gamma_d, nx, ny);
    else if(dimPoint==3 && dimVect==1)
        GaussGpuGrad1ConvOnDevice<float,3,1><<<gridSize,blockSize,blockSize.x*(dimPoint+dimVect)*sizeof(float)>>>
        (ooSigma2, alpha_d, x_d, y_d, beta_d, gamma_d, nx, ny);
    else if(dimPoint==3 && dimVect==3)
        GaussGpuGrad1ConvOnDevice<float,3,3><<<gridSize,blockSize,blockSize.x*(dimPoint+dimVect)*sizeof(float)>>>
        (ooSigma2, alpha_d, x_d, y_d, beta_d, gamma_d, nx, ny);
    else
    {
        printf("error: dimensions of Gauss kernel not implemented in cuda");
    hipFree(x_d);
    hipFree(y_d);
    hipFree(alpha_d);
    hipFree(gamma_d);
    hipFree(beta_d);
        return(-1);
    }

    // block until the device has completed
    hipDeviceSynchronize();

    // Send data from device to host.
    hipMemcpy(gamma_h, gamma_d, sizeof(float)*(nx*dimPoint),hipMemcpyDeviceToHost);

    // Free memory.
    hipFree(x_d);
    hipFree(y_d);
    hipFree(alpha_d);
    hipFree(gamma_d);
    hipFree(beta_d);

    return 0;
}


//////////////////////////////////////////////////////////////

#if UseCudaOnDoubles  
extern "C" int GaussGpuGrad1Conv_double(double ooSigma2,
                                       double* alpha_h, double* x_h, double* y_h, double* beta_h, double* gamma_h,
                                       int dimPoint, int dimVect, int nx, int ny)
{

    // Data on the device.
    double* x_d;
    double* y_d;
    double* alpha_d;
    double* gamma_d;
    double* beta_d;

    // Allocate arrays on device.
    hipMalloc((void**)&x_d, sizeof(double)*(nx*dimPoint));
    hipMalloc((void**)&y_d, sizeof(double)*(ny*dimPoint));
    hipMalloc((void**)&alpha_d, sizeof(double)*(nx*dimVect));
    hipMalloc((void**)&beta_d, sizeof(double)*(ny*dimVect));
    hipMalloc((void**)&gamma_d, sizeof(double)*(nx*dimPoint));

    // Send data from host to device.
    hipMemcpy(x_d, x_h, sizeof(double)*(nx*dimPoint), hipMemcpyHostToDevice);
    hipMemcpy(y_d, y_h, sizeof(double)*(ny*dimPoint), hipMemcpyHostToDevice);
    hipMemcpy(alpha_d, alpha_h, sizeof(double)*(nx*dimVect), hipMemcpyHostToDevice);
    hipMemcpy(beta_d, beta_h, sizeof(double)*(ny*dimVect), hipMemcpyHostToDevice);

    // compute on device.
    dim3 blockSize;
    blockSize.x = CUDA_BLOCK_SIZE; // number of threads in each block
    dim3 gridSize;
    gridSize.x =  nx / blockSize.x + (nx%blockSize.x==0 ? 0 : 1);

    if(dimPoint==1 && dimVect==1)
        GaussGpuGrad1ConvOnDevice<double,1,1><<<gridSize,blockSize,blockSize.x*(dimPoint+dimVect)*sizeof(double)>>>
        (ooSigma2, alpha_d, x_d, y_d, beta_d, gamma_d, nx, ny);
    else if(dimPoint==2 && dimVect==1)
        GaussGpuGrad1ConvOnDevice<double,2,1><<<gridSize,blockSize,blockSize.x*(dimPoint+dimVect)*sizeof(double)>>>
        (ooSigma2, alpha_d, x_d, y_d, beta_d, gamma_d, nx, ny);
    else if(dimPoint==2 && dimVect==2)
        GaussGpuGrad1ConvOnDevice<double,2,2><<<gridSize,blockSize,blockSize.x*(dimPoint+dimVect)*sizeof(double)>>>
        (ooSigma2, alpha_d, x_d, y_d, beta_d, gamma_d, nx, ny);
    else if(dimPoint==3 && dimVect==1)
        GaussGpuGrad1ConvOnDevice<double,3,1><<<gridSize,blockSize,blockSize.x*(dimPoint+dimVect)*sizeof(double)>>>
        (ooSigma2, alpha_d, x_d, y_d, beta_d, gamma_d, nx, ny);
    else if(dimPoint==3 && dimVect==3)
        GaussGpuGrad1ConvOnDevice<double,3,3><<<gridSize,blockSize,blockSize.x*(dimPoint+dimVect)*sizeof(double)>>>
        (ooSigma2, alpha_d, x_d, y_d, beta_d, gamma_d, nx, ny);
    else
    {
        printf("error: dimensions of Gauss kernel not implemented in cuda");
    hipFree(x_d);
    hipFree(y_d);
    hipFree(alpha_d);
    hipFree(gamma_d);
    hipFree(beta_d);
        return(-1);
    }

    // block until the device has completed
    hipDeviceSynchronize();

    // Send data from device to host.
    hipMemcpy(gamma_h, gamma_d, sizeof(double)*(nx*dimPoint),hipMemcpyDeviceToHost);

    // Free memory.
    hipFree(x_d);
    hipFree(y_d);
    hipFree(alpha_d);
    hipFree(gamma_d);
    hipFree(beta_d);

    return 0;
}
#endif

void ExitFcn(void)
{
  hipDeviceReset();
}


//////////////////////////////////////////////////////////////////
///////////////// MEX ENTRY POINT ////////////////////////////////
//////////////////////////////////////////////////////////////////

 
 /* the gateway function */
 void mexFunction( int nlhs, mxArray *plhs[],
                   int nrhs, const mxArray *prhs[])
 //plhs: double *gamma
 //prhs: double *alpha, double *x, double *y, double *beta, double sigma
 
 { 
   // register an exit function to prevent crash at matlab exit or recompiling
   mexAtExit(ExitFcn);

   /*  check for proper number of arguments */
   if(nrhs != 5) 
     mexErrMsgTxt("5 inputs required.");
   if(nlhs < 1 | nlhs > 1) 
     mexErrMsgTxt("One output required.");
 
   //////////////////////////////////////////////////////////////
   // Input arguments
   //////////////////////////////////////////////////////////////
   
   int argu = -1;
 
   //------ the first input argument: alpha---------------//
   argu++;
   /*  create a pointer to the input vectors wts */
   double *alpha = mxGetPr(prhs[argu]);
   /*  get the dimensions of the input weights */
   int dimvect = mxGetM(prhs[argu]);
   int nx = mxGetN(prhs[argu]); //ncols
  
   //----- the second input argument: x--------------//
   argu++;
   /*  create a pointer to the input vectors srcs */
   double *x = mxGetPr(prhs[argu]);
   /*  input sources */
   int dimpoint = mxGetM(prhs[argu]); //mrows
   /* check to make sure the number of columns is nx */
   if( mxGetN(prhs[argu])!=nx ) {
     mexErrMsgTxt("Input x must have same number of columns as alpha.");
   }
  
   //----- the third input argument: y--------------//
   argu++;
   /*  create a pointer to the input vectors tgts */
   double *y = mxGetPr(prhs[argu]);
   /*  input sources */
   int ny = mxGetN(prhs[argu]); //ncols
   /* check to make sure the number of rows is dimpoint */
   if( mxGetM(prhs[argu])!=dimpoint )
     mexErrMsgTxt("Input y must have same number of rows as x.");
 
   //------ the fourth input argument: beta---------------//
   argu++;
   /*  create a pointer to the input vectors wts */
   double *beta = mxGetPr(prhs[argu]);
   /* check to make sure the number of rows is dimvect */
   if( mxGetM(prhs[argu])!=dimvect )
     mexErrMsgTxt("Input y must have same number of rows as alpha.");
   /* check to make sure the number of columns is ny */
   if( mxGetN(prhs[argu])!=ny )
     mexErrMsgTxt("Input beta must have same number of columns as y.");

   //----- the fifth input argument: sigma-------------//
   argu++;
   /* check to make sure the input argument is a scalar */
   if( !mxIsDouble(prhs[argu]) || mxIsComplex(prhs[argu]) ||
       mxGetN(prhs[argu])*mxGetM(prhs[argu])!=1 ) {
     mexErrMsgTxt("Input sigma must be a scalar.");
   }
   /*  get the scalar input sigma */
   double sigma = mxGetScalar(prhs[argu]);
   if (sigma <= 0.0)
 	  mexErrMsgTxt("Input sigma must be a positive number.");
   double oosigma2 = 1.0f/(sigma*sigma);
 
   //////////////////////////////////////////////////////////////
   // Output arguments
   //////////////////////////////////////////////////////////////
   /*  set the output pointer to the output result(vector) */
   plhs[0] = mxCreateDoubleMatrix(dimpoint,nx,mxREAL);
   
   /*  create a C pointer to a copy of the output result(vector)*/
   double *gamma = mxGetPr(plhs[0]);
   
#if UseCudaOnDoubles
   GaussGpuGrad1Conv_double(oosigma2,alpha,x,y,beta,gamma,dimpoint,dimvect,nx,ny);  
#else
   // convert to float
   float *alpha_f = new float[nx*dimvect];
   float *x_f = new float[nx*dimpoint];
   float *y_f = new float[ny*dimpoint];
   float *beta_f = new float[ny*dimvect];
   float *gamma_f = new float[nx*dimpoint];
   for(int i=0; i<nx*dimvect; i++)
     alpha_f[i] = alpha[i];
   for(int i=0; i<nx*dimpoint; i++)
     x_f[i] = x[i];
   for(int i=0; i<ny*dimpoint; i++)
     y_f[i] = y[i];
   for(int i=0; i<ny*dimvect; i++)
     beta_f[i] = beta[i];
   
   // function calls;
   GaussGpuGrad1Conv_float(oosigma2,alpha_f,x_f,y_f,beta_f,gamma_f,dimpoint,dimvect,nx,ny);
 
   for(int i=0; i<nx*dimpoint; i++)
       gamma[i] = gamma_f[i];

   delete [] alpha_f;
   delete [] x_f;
   delete [] y_f;
   delete [] beta_f;
   delete [] gamma_f;
#endif

   return;
   
 }



